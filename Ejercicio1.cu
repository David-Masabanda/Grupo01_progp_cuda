#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <fstream>
#include <vector>
#include <random>
#include <fmt/core.h>
#include <cmath>
#include <chrono>
namespace ch=std::chrono;

#define GRUPOS 10

std::vector<int> read_file() {
    std::fstream fs("C:/Users/josue/Desktop/Universidad/ProgParalela/Grupal_final/numeros/datos2.txt", std::ios::in);
    std::string line;
    std::vector<int> ret;
    while (std::getline(fs, line)) {
        ret.push_back(std::stoi(line));
    }
    fs.close();
    return ret;
}

__global__ void histograma(int *histo, int *data, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        atomicAdd(&histo[data[idx]], 1);
    }
}

int main() {

    std::vector<int> data = read_file();
    int n = data.size();
    int block = std::ceil((double)255 / GRUPOS);

    std::vector<int> histo(256);
    std::vector<int> final(GRUPOS);

    // Definir punteros para la GPU
    int *d_histo;
    int *d_data;

    hipMalloc((void**)&d_data, n * sizeof(int));
    hipMalloc((void**)&d_histo, 256 * sizeof(int));

    // Copiar datos desde la CPU a la GPU
    auto start = ch::high_resolution_clock::now();
    hipMemcpy(d_data, data.data(), n * sizeof(int), hipMemcpyHostToDevice);

    // Llamar al kernel
    histograma<<<(n + 255) / 256, 256>>>(d_histo, d_data, n);

    // Copiar el histograma resultante de la GPU a la CPU
    hipMemcpy(histo.data(), d_histo, 256 * sizeof(int), hipMemcpyDeviceToHost);

    // Liberar memoria en la GPU
    hipFree(d_data);
    hipFree(d_histo);

    int indice= 0;
    for (int i = 0; i < GRUPOS; ++i) {
        int suma = 0;
        for (int j = 0; j < block; ++j) {
            if (indice<256){
                suma += histo[indice++];
            }
        }
        final[i] = suma;
        fmt::println("[{}], [{}]", i, suma);
    }

    auto end = std::chrono::high_resolution_clock::now();
    ch::duration<double, std::milli> tiempo = end - start;
    fmt::println("Tiempo CUDA: {}ms", tiempo.count());

    return 0;
}