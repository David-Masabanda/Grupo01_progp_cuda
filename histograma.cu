#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <fstream>
#include <vector>
#include <random>
#include <fmt/core.h>
#include <chrono>
namespace ch=std::chrono;

#define NCLASES 256
#define BS 256

std::vector<int> read_file() {
    std::fstream fs("C:/Users/josue/Desktop/Universidad/ProgParalela/Grupal_final/datos4.txt", std::ios::in);
    std::string line;
    std::vector<int> ret;
    while (std::getline(fs, line)) {
        ret.push_back(std::stoi(line));
    }
    fs.close();
    return ret;
}

__global__ void histo_k(int *histo, int *data, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n)
        atomicAdd(&histo[data[i]], 1);
}

int main() {
    // Generar datos aleatorios
    std::vector<int> data = read_file();
    int n = data.size();

    // Inicializar arreglo para el histograma
    int histo[NCLASES] = {0};

    // Definir punteros para la GPU
    int *d_histo;
    int *d_data;
    hipMalloc((void**)&d_data, n * sizeof(int));
    hipMalloc((void**)&d_histo, NCLASES * sizeof(int));

    // Copiar datos desde la CPU a la GPU
    auto start = ch::high_resolution_clock::now();

    hipMemcpy(d_data, data.data(), n * sizeof(int), hipMemcpyHostToDevice);

    // Llamar al kernel
    histo_k<<<(n + BS - 1) / BS, BS>>>(d_histo, d_data, n);

    // Copiar el histograma resultante de la GPU a la CPU
    hipMemcpy(histo, d_histo, NCLASES * sizeof(int), hipMemcpyDeviceToHost);


    auto end = std::chrono::high_resolution_clock::now();
    ch::duration<double, std::milli> tiempo = end-start;
    fmt::println("Tiempo CUDA: {}ms", tiempo.count());

    // Liberar memoria en la GPU
    hipFree(d_data);
    hipFree(d_histo);

    // Imprimir el histograma resultante
    for (int i = 0; i < NCLASES; ++i) {
        std::cout << "histo[" << i << "]: " << histo[i] << std::endl;
    }

    return 0;
}